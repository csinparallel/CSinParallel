#include <stdio.h>
#include <hip/hip_runtime.h>

// !!!!!! NOTE:
//  NVIDIA refers to these functions prefaced with __global__ 
//  as 'kernel' functions that run on the GPU 'device'.
__global__ void hello() {
    // special dim3 variables available to each thread in a kernel
    // or device function:
    // blockIdx    the x, y, z coordinate of the block in the grid
    // threadIdX   the x, y, z coordinate of the thread in the block
    printf("I am thread (%d, %d, %d) of block (%d, %d, %d) in the grid\n",
           threadIdx.x, threadIdx.y, threadIdx.z, 
           blockIdx.x, blockIdx.y, blockIdx.z );
}

// Note that this is called from the host, not the GPU device.
// We create dim3 structs there and can print their components
// with this function.
void printDims(dim3 gridDim, dim3 blockDim) {
    printf("Grid Dimensions : [%d, %d, %d] blocks. \n",
    gridDim.x, gridDim.y, gridDim.z);

    printf("Block Dimensions : [%d, %d, %d] threads.\n",
    blockDim.x, blockDim.y, blockDim.z);
}

int main(int argc, char **argv) {

    // dim3 is a special data type: a vector of 3 integers.
    // each integer is accessed using .x, .y and .z 
    // (see printDims() above)

    // 1 dimensionsional case is the following: 1D grid of 1D block
    dim3 gridDim(1);      // 1 blocks in x direction, y, z default to 1
    dim3 blockDim(8);     // 8 threads per block in x direction

    // TODO: Try 128 threads in a block. What do you observe?
    //       Try the maximum threads per block allowed for your card.
    //       See device_info example.
    //       Try over the maximum threads per block for your card.
   
    printDims(gridDim, blockDim);
    
    printf("From each thread:\n");
    hello<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();     // need for printfs in kernel to flush

    return 0;
}
